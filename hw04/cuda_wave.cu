/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */

float *hostVal;
float *devOldVal, *devNewVal;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
    char tchar[20];

    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: "
                ,MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n", 
                    MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }

    printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__ 
void init_line(int __tpoints, float* __oldval, float* __newval)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    /* Calculate initial values based on sine curve */
    for (int i = index; i < __tpoints; i+=stride) {
        float x = (float)i / (__tpoints-1);
        __oldval[i] = __newval[i] = __sinf(2.0 * PI * x);
    } 
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__device__
float do_math(float __newval, float __oldval)
{
    float dtime, c, dx, tau, sqtau;

    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;
    return (2.0 * __newval) - __oldval + (sqtau *  (-2.0)*__newval);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__
void update(int __tpoints, int __nsteps, float* __oldval, float* __newval)
{
    float __localval;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    /* Update values for each time step */
    for (int i = 0; i< __nsteps; i++) {
        /* Update points along line for this time step */
        for (int j = index; j < __tpoints; j+=stride) {
            /* global endpoints */
            if ((j == 0) || (j  == (__tpoints-1)))
                __newval[j] = 0.0;
            else
                __localval = do_math(__newval[j], __oldval[j]);
                __oldval[j] = __newval[j];
                __newval[j] = __localval;
        }
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
    int i;

    for (i = 0; i < tpoints; i++) {
        printf("%6.4f ", hostVal[i]);
        if ((i+1)%10 == 0)
            printf("\n");
    }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);

    int blockSize = 256;
    int numBlocks = (tpoints + blockSize + 1) / blockSize; 

    hipMalloc(&devOldVal, (MAXPOINTS+2) * sizeof(float));
    hipMalloc(&devNewVal, (MAXPOINTS+2) * sizeof(float));

    check_param();
    printf("Initializing points on the line...\n");
    init_line<<<numBlocks, blockSize>>>(tpoints, devOldVal, devNewVal);
    printf("Updating all points for all time steps...\n");
    update<<<numBlocks, blockSize>>>(tpoints, nsteps, devOldVal, devNewVal);

    hostVal = (float*) malloc(sizeof(float) * tpoints);

    hipDeviceSynchronize();

    hipMemcpy(hostVal, devNewVal, sizeof(float) * tpoints, hipMemcpyDeviceToHost);

    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");

    return 0;
}
